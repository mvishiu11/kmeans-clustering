#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <limits>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cfloat>

using namespace std;

/**
 * CUDA kernel to assign points to the nearest cluster.
 *
 * @param points Array of points.
 * @param centroids Array of centroids.
 * @param cluster_assignments Output array for cluster assignments.
 * @param N Number of points.
 * @param n Dimensionality of each point.
 * @param k Number of clusters.
 */
__global__ void assign_clusters(const float* points, float* centroids, int* cluster_assignments, int N, int n, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float min_distance = FLT_MAX;
        int best_cluster = -1;

        for (int cluster = 0; cluster < k; ++cluster) {
            float distance = 0.0f;

            for (int dim = 0; dim < n; ++dim) {
                float diff = points[idx * n + dim] - centroids[cluster * n + dim];
                distance += diff * diff;
            }

            if (distance < min_distance) {
                min_distance = distance;
                best_cluster = cluster;
            }
        }

        cluster_assignments[idx] = best_cluster;
    }
}

/**
 * CUDA kernel to compute new centroids by aggregating points in each cluster.
 *
 * @param points Array of points.
 * @param centroids Output array for new centroids.
 * @param cluster_assignments Array of cluster assignments for each point.
 * @param cluster_sizes Output array for the number of points in each cluster.
 * @param N Number of points.
 * @param n Dimensionality of each point.
 * @param k Number of clusters.
 */
__global__ void compute_centroids(const float* points, float* centroids, const int* cluster_assignments, int* cluster_sizes, int N, int n, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        int cluster = cluster_assignments[idx];

        for (int dim = 0; dim < n; ++dim) {
            atomicAdd(&centroids[cluster * n + dim], points[idx * n + dim]);
        }

        atomicAdd(&cluster_sizes[cluster], 1);
    }
}

/**
 * Initialize centroids randomly from the dataset.
 *
 * @param centroids Output vector to store initial centroids.
 * @param points Input vector of points.
 * @param N Number of points.
 * @param n Dimensionality of each point.
 * @param k Number of clusters.
 */
void initialize_centroids(vector<float>& centroids, const vector<float>& points, int N, int n, int k) {
    srand(42);
    for (int i = 0; i < k; ++i) {
        int random_index = rand() % N;
        for (int dim = 0; dim < n; ++dim) {
            centroids[i * n + dim] = points[random_index * n + dim];
        }
    }
}

/**
 * Main function to run the k-means clustering algorithm.
 *
 * @param argc Number of command-line arguments.
 * @param argv Array of command-line arguments.
 * @return Exit status.
 */
int main(int argc, char** argv) {
    if (argc != 5) {
        cerr << "Usage: " << argv[0] << " <input_file> <output_file> <k> <max_iters>\n";
        return 1;
    }

    string input_file = argv[1];
    string output_file = argv[2];
    int k = stoi(argv[3]);
    int max_iters = stoi(argv[4]);

    // Read input file
    ifstream infile(input_file);
    if (!infile) {
        cerr << "Error: Unable to open input file." << endl;
        return 1;
    }

    int N, n;
    infile >> N >> n;

    vector<float> points(N * n);
    for (int i = 0; i < N * n; ++i) {
        infile >> points[i];
    }

    infile.close();

    // Allocate memory for centroids and initialize
    vector<float> centroids(k * n, 0.0f);
    initialize_centroids(centroids, points, N, n, k);

    // Device memory allocations
    float* d_points;
    float* d_centroids;
    int* d_cluster_assignments;
    int* d_cluster_sizes;

    hipMalloc(&d_points, N * n * sizeof(float));
    hipMalloc(&d_centroids, k * n * sizeof(float));
    hipMalloc(&d_cluster_assignments, N * sizeof(int));
    hipMalloc(&d_cluster_sizes, k * sizeof(int));

    hipMemcpy(d_points, points.data(), N * n * sizeof(float), hipMemcpyHostToDevice);

    // Iterative k-means computation
    for (int iter = 0; iter < max_iters; ++iter) {
        hipMemcpy(d_centroids, centroids.data(), k * n * sizeof(float), hipMemcpyHostToDevice);

        int threads = 1024;
        int blocks = (N + threads - 1) / threads;
        assign_clusters<<<blocks, threads>>>(d_points, d_centroids, d_cluster_assignments, N, n, k);

        hipMemset(d_centroids, 0, k * n * sizeof(float));
        hipMemset(d_cluster_sizes, 0, k * sizeof(int));
        compute_centroids<<<blocks, threads>>>(d_points, d_centroids, d_cluster_assignments, d_cluster_sizes, N, n, k);

        vector<float> new_centroids(k * n, 0.0f);
        vector<int> cluster_sizes(k, 0);
        hipMemcpy(new_centroids.data(), d_centroids, k * n * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(cluster_sizes.data(), d_cluster_sizes, k * sizeof(int), hipMemcpyDeviceToHost);

        for (int cluster = 0; cluster < k; ++cluster) {
            if (cluster_sizes[cluster] > 0) {
                for (int dim = 0; dim < n; ++dim) {
                    new_centroids[cluster * n + dim] /= cluster_sizes[cluster];
                }
            }
        }

        centroids = new_centroids;
    }

    vector<int> cluster_assignments(N);
    hipMemcpy(cluster_assignments.data(), d_cluster_assignments, N * sizeof(int), hipMemcpyDeviceToHost);

    // Write output file
    ofstream outfile(output_file);
    if (!outfile) {
        cerr << "Error: Unable to open output file." << endl;
        return 1;
    }

    for (int i = 0; i < N; ++i) {
        for (int dim = 0; dim < n; ++dim) {
            outfile << points[i * n + dim] << " ";
        }
        outfile << cluster_assignments[i] << "\n";
    }

    outfile.close();

    // Clean up device memory
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_cluster_assignments);
    hipFree(d_cluster_sizes);

    return 0;
}